/*!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
! Copyright 2010.  Los Alamos National Security, LLC. This material was    !
! produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos !
! National Laboratory (LANL), which is operated by Los Alamos National     !
! Security, LLC for the U.S. Department of Energy. The U.S. Government has !
! rights to use, reproduce, and distribute this software.  NEITHER THE     !
! GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY,     !
! EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS         !
! SOFTWARE.  If software is modified to produce derivative works, such     !
! modified software should be clearly marked, so as not to confuse it      !
! with the version available from LANL.                                    !
!                                                                          !
! Additionally, this program is free software; you can redistribute it     !
! and/or modify it under the terms of the GNU General Public License as    !
! published by the Free Software Foundation; version 2.0 of the License.   !
! Accordingly, this program is distributed in the hope that it will be     !
! useful, but WITHOUT ANY WARRANTY; without even the implied warranty of   !
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General !
! Public License for more details.                                         !
!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!*/

#include "Matrix.h"

extern hipsparseHandle_t* cshandle;
extern int ndevices;
extern int nblocks;

// Multiply matrices in csr format - A * B = C
void M_SparseMatrixMultiply(SparseMatrix &A, SparseMatrix &B, SparseMatrix &C) {

  int cdev;

  // Get current device
  hipGetDevice(&cdev);

  // On each GPU
  for (int d = 0; d < ndevices; d++) {

    hipSetDevice(d);

    // Determine number of non-zeros for multiplication - C = A * B
    hipsparseXcsrgemmNnz(cshandle[d], 
        HIPSPARSE_OPERATION_NON_TRANSPOSE, 
        HIPSPARSE_OPERATION_NON_TRANSPOSE, A.N, A.M, A.K,
        A.descr, A.nnz, A.csrRowPtr, A.csrColInd,
        B.descr, B.nnz, B.csrRowPtr, B.csrColInd,
        C.descr, C.csrRowPtr, C.nnzTotalDevHostPtr);

    C.nnz = *C.nnzTotalDevHostPtr;

#if REALSIZE==4
    hipsparseScsrgemm(cshandle[d],
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE, A.N, A.M, A.K,
        A.descr, A.nnz,
        A.csrVal, A.csrRowPtr, A.csrColInd,
        B.descr, B.nnz,
        B.csrVal, B.csrRowPtr, B.csrColInd,
        C.descr,
        C.csrVal, C.csrRowPtr, C.csrColInd);
#elif REALSIZE==8
    hipsparseDcsrgemm(cshandle[d], 
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE, A.N, A.M, A.K,
        A.descr, A.nnz,
        A.csrVal, A.csrRowPtr, A.csrColInd,
        B.descr, B.nnz,
        B.csrVal, B.csrRowPtr, B.csrColInd,
        C.descr,
        C.csrVal, C.csrRowPtr, C.csrColInd);
#endif
  }

  // Restore device
  hipSetDevice(cdev);

}
