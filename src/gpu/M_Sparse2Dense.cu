/*!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
! Copyright 2010.  Los Alamos National Security, LLC. This material was    !
! produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos !
! National Laboratory (LANL), which is operated by Los Alamos National     !
! Security, LLC for the U.S. Department of Energy. The U.S. Government has !
! rights to use, reproduce, and distribute this software.  NEITHER THE     !
! GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY,     !
! EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS         !
! SOFTWARE.  If software is modified to produce derivative works, such     !
! modified software should be clearly marked, so as not to confuse it      !
! with the version available from LANL.                                    !
!                                                                          !
! Additionally, this program is free software; you can redistribute it     !
! and/or modify it under the terms of the GNU General Public License as    !
! published by the Free Software Foundation; version 2.0 of the License.   !
! Accordingly, this program is distributed in the hope that it will be     !
! useful, but WITHOUT ANY WARRANTY; without even the implied warranty of   !
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General !
! Public License for more details.                                         !
!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!*/

#include "Matrix.h"

extern hipsparseHandle_t* cshandle;
extern int ndevices;
extern int nblocks;

// Convert sparse matrix in csr format to dense matrix 
void M_Sparse2Dense(SparseMatrix &A, Matrix &B) {

  int cdev;

  // Get current device
  hipGetDevice(&cdev);

  // On each GPU
  for (int d = 0; d < ndevices; d++) {

    hipSetDevice(d);

#if REALSIZE==4
    hipsparseScsr2dense(cshandle[d],
      A.N, A.M, A.descr, A.csrVal, A.csrRowPtr, A.csrColInd,
      B.Device[d], B.DN);
#elif REALSIZE==8
    hipsparseDcsr2dense(cshandle[d], 
      A.N, A.M, A.descr, A.csrVal, A.csrRowPtr, A.csrColInd, 
      B.Device[d], B.DN);
#endif

  }

  // Restore device
  hipSetDevice(cdev);

}
